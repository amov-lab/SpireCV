#include "hip/hip_runtime.h"
#include "yololayer.h"
#include "cuda_utils.h"

#include <cassert>
#include <vector>
#include <iostream>

namespace Tn {
template<typename T> 
void write(char*& buffer, const T& val) {
  *reinterpret_cast<T*>(buffer) = val;
  buffer += sizeof(T);
}

template<typename T> 
void read(const char*& buffer, T& val) {
  val = *reinterpret_cast<const T*>(buffer);
  buffer += sizeof(T);
}
}

namespace nvinfer1 {
YoloLayerPlugin::YoloLayerPlugin(int classCount, int netWidth, int netHeight, int maxOut, bool is_segmentation, const std::vector<YoloKernel>& vYoloKernel) {
  mClassCount = classCount;
  mYoloV5NetWidth = netWidth;
  mYoloV5NetHeight = netHeight;
  mMaxOutObject = maxOut;
  is_segmentation_ = is_segmentation;
  mYoloKernel = vYoloKernel;
  mKernelCount = vYoloKernel.size();

  CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
  size_t AnchorLen = sizeof(float)* kNumAnchor * 2;
  for (int ii = 0; ii < mKernelCount; ii++) {
    CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
    const auto& yolo = mYoloKernel[ii];
    CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
  }
}

YoloLayerPlugin::~YoloLayerPlugin() {
  for (int ii = 0; ii < mKernelCount; ii++) {
    CUDA_CHECK(hipFree(mAnchor[ii]));
  }
  CUDA_CHECK(hipHostFree(mAnchor));
}

// create the plugin at runtime from a byte stream
YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length) {
  using namespace Tn;
  const char *d = reinterpret_cast<const char *>(data), *a = d;
  read(d, mClassCount);
  read(d, mThreadCount);
  read(d, mKernelCount);
  read(d, mYoloV5NetWidth);
  read(d, mYoloV5NetHeight);
  read(d, mMaxOutObject);
  read(d, is_segmentation_);
  mYoloKernel.resize(mKernelCount);
  auto kernelSize = mKernelCount * sizeof(YoloKernel);
  memcpy(mYoloKernel.data(), d, kernelSize);
  d += kernelSize;
  CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
  size_t AnchorLen = sizeof(float)* kNumAnchor * 2;
  for (int ii = 0; ii < mKernelCount; ii++) {
    CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
    const auto& yolo = mYoloKernel[ii];
    CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
  }
  assert(d == a + length);
}

void YoloLayerPlugin::serialize(void* buffer) const TRT_NOEXCEPT {
  using namespace Tn;
  char* d = static_cast<char*>(buffer), *a = d;
  write(d, mClassCount);
  write(d, mThreadCount);
  write(d, mKernelCount);
  write(d, mYoloV5NetWidth);
  write(d, mYoloV5NetHeight);
  write(d, mMaxOutObject);
  write(d, is_segmentation_);
  auto kernelSize = mKernelCount * sizeof(YoloKernel);
  memcpy(d, mYoloKernel.data(), kernelSize);
  d += kernelSize;

  assert(d == a + getSerializationSize());
}

size_t YoloLayerPlugin::getSerializationSize() const TRT_NOEXCEPT {
  size_t s = sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount);
  s += sizeof(YoloKernel) * mYoloKernel.size();
  s += sizeof(mYoloV5NetWidth) + sizeof(mYoloV5NetHeight);
  s += sizeof(mMaxOutObject) + sizeof(is_segmentation_);
  return s;
}

int YoloLayerPlugin::initialize() TRT_NOEXCEPT {
  return 0;
}

Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims) TRT_NOEXCEPT {
  //output the result to channel
  int totalsize = mMaxOutObject * sizeof(Detection) / sizeof(float);
  return Dims3(totalsize + 1, 1, 1);
}

// Set plugin namespace
void YoloLayerPlugin::setPluginNamespace(const char* pluginNamespace) TRT_NOEXCEPT {
  mPluginNamespace = pluginNamespace;
}

const char* YoloLayerPlugin::getPluginNamespace() const TRT_NOEXCEPT {
  return mPluginNamespace;
}

// Return the DataType of the plugin output at the requested index
DataType YoloLayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const TRT_NOEXCEPT {
  return DataType::kFLOAT;
}

// Return true if output tensor is broadcast across a batch.
bool YoloLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const TRT_NOEXCEPT {
  return false;
}

// Return true if plugin can use input that is broadcast across batch without replication.
bool YoloLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const TRT_NOEXCEPT {
  return false;
}

void YoloLayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput) TRT_NOEXCEPT {}

// Attach the plugin object to an execution context and grant the plugin the access to some context resource.
void YoloLayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator) TRT_NOEXCEPT {}

// Detach the plugin object from its execution context.
void YoloLayerPlugin::detachFromContext() TRT_NOEXCEPT {}

const char* YoloLayerPlugin::getPluginType() const TRT_NOEXCEPT {
  return "YoloLayer_TRT";
}

const char* YoloLayerPlugin::getPluginVersion() const TRT_NOEXCEPT {
  return "1";
}

void YoloLayerPlugin::destroy() TRT_NOEXCEPT {
  delete this;
}

// Clone the plugin
IPluginV2IOExt* YoloLayerPlugin::clone() const TRT_NOEXCEPT {
  YoloLayerPlugin* p = new YoloLayerPlugin(mClassCount, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, is_segmentation_, mYoloKernel);
  p->setPluginNamespace(mPluginNamespace);
  return p;
}

__device__ float Logist(float data) { return 1.0f / (1.0f + expf(-data)); };

__global__ void CalDetection(const float *input, float *output, int noElements,
    const int netwidth, const int netheight, int maxoutobject, int yoloWidth,
    int yoloHeight, const float anchors[kNumAnchor * 2], int classes, int outputElem, bool is_segmentation) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= noElements) return;

  int total_grid = yoloWidth * yoloHeight;
  int bnIdx = idx / total_grid;
  idx = idx - total_grid * bnIdx;
  int info_len_i = 5 + classes;
  if (is_segmentation) info_len_i += 32;
  const float* curInput = input + bnIdx * (info_len_i * total_grid * kNumAnchor);

  for (int k = 0; k < kNumAnchor; ++k) {
    float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
    if (box_prob < kIgnoreThresh) continue;
    int class_id = 0;
    float max_cls_prob = 0.0;
    for (int i = 5; i < 5 + classes; ++i) {
      float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
      if (p > max_cls_prob) {
        max_cls_prob = p;
        class_id = i - 5;
      }
    }
    float *res_count = output + bnIdx * outputElem;
    int count = (int)atomicAdd(res_count, 1);
    if (count >= maxoutobject) return;
    char *data = (char*)res_count + sizeof(float) + count * sizeof(Detection);
    Detection *det = (Detection*)(data);

    int row = idx / yoloWidth;
    int col = idx % yoloWidth;

    det->bbox[0] = (col - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * netwidth / yoloWidth;
    det->bbox[1] = (row - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * netheight / yoloHeight;

    det->bbox[2] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]);
    det->bbox[2] = det->bbox[2] * det->bbox[2] * anchors[2 * k];
    det->bbox[3] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]);
    det->bbox[3] = det->bbox[3] * det->bbox[3] * anchors[2 * k + 1];
    det->conf = box_prob * max_cls_prob;
    det->class_id = class_id;

    for (int i = 0; is_segmentation && i < 32; i++) {
      det->mask[i] = curInput[idx + k * info_len_i * total_grid + (i + 5 + classes) * total_grid];
    }
  }
}

void YoloLayerPlugin::forwardGpu(const float* const* inputs, float *output, hipStream_t stream, int batchSize) {
  int outputElem = 1 + mMaxOutObject * sizeof(Detection) / sizeof(float);
  for (int idx = 0; idx < batchSize; ++idx) {
    CUDA_CHECK(hipMemsetAsync(output + idx * outputElem, 0, sizeof(float), stream));
  }
  int numElem = 0;
  for (unsigned int i = 0; i < mYoloKernel.size(); ++i) {
    const auto& yolo = mYoloKernel[i];
    numElem = yolo.width * yolo.height * batchSize;
    if (numElem < mThreadCount) mThreadCount = numElem;

    CalDetection << < (numElem + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream >> >
      (inputs[i], output, numElem, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, yolo.width, yolo.height, (float*)mAnchor[i], mClassCount, outputElem, is_segmentation_);
  }
}


int YoloLayerPlugin::enqueue(int batchSize, const void* const* inputs, void* TRT_CONST_ENQUEUE* outputs, void* workspace, hipStream_t stream) TRT_NOEXCEPT {
  forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, batchSize);
  return 0;
}

PluginFieldCollection YoloPluginCreator::mFC{};
std::vector<PluginField> YoloPluginCreator::mPluginAttributes;

YoloPluginCreator::YoloPluginCreator() {
  mPluginAttributes.clear();
  mFC.nbFields = mPluginAttributes.size();
  mFC.fields = mPluginAttributes.data();
}

const char* YoloPluginCreator::getPluginName() const TRT_NOEXCEPT {
  return "YoloLayer_TRT";
}

const char* YoloPluginCreator::getPluginVersion() const TRT_NOEXCEPT {
  return "1";
}

const PluginFieldCollection* YoloPluginCreator::getFieldNames() TRT_NOEXCEPT {
  return &mFC;
}

IPluginV2IOExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) TRT_NOEXCEPT {
  assert(fc->nbFields == 2);
  assert(strcmp(fc->fields[0].name, "netinfo") == 0);
  assert(strcmp(fc->fields[1].name, "kernels") == 0);
  int *p_netinfo = (int*)(fc->fields[0].data);
  int class_count = p_netinfo[0];
  int input_w = p_netinfo[1];
  int input_h = p_netinfo[2];
  int max_output_object_count = p_netinfo[3];
  bool is_segmentation = (bool)p_netinfo[4];
  std::vector<YoloKernel> kernels(fc->fields[1].length);
  memcpy(&kernels[0], fc->fields[1].data, kernels.size() * sizeof(YoloKernel));
  YoloLayerPlugin* obj = new YoloLayerPlugin(class_count, input_w, input_h, max_output_object_count, is_segmentation, kernels);
  obj->setPluginNamespace(mNamespace.c_str());
  return obj;
}

IPluginV2IOExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) TRT_NOEXCEPT {
  // This object will be deleted when the network is destroyed, which will
  // call YoloLayerPlugin::destroy()
  YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
  obj->setPluginNamespace(mNamespace.c_str());
  return obj;
}
}

